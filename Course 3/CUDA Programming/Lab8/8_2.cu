
#include <hip/hip_runtime.h>
#include <iostream>
#include <hipfft/hipfft.h>
#include <malloc.h>
#include <fstream>

#define NX 365
#define BATCH 1

int main()
{
    hipfftHandle plan;
    hipfftComplex *data;
    hipfftComplex *data_h = (hipfftComplex *)calloc(NX * BATCH, sizeof(hipfftComplex));

    std::ifstream fin("data.txt");
    if(fin.fail())
    {
        std::cerr << "Failed to open file." << std::endl;
        return -1;
    }
    std::ofstream fout("results.txt");
    float month, day, wolf, temp;
    for(int i = 0; i < NX; i++)
    {
        fin >> month >> day >> wolf >> temp;
        if(wolf != 999)
            data_h[i].x = wolf;
        else
            data_h[i].x = 0.0f;
        data_h[i].y = 0.0f;
    }

    hipMalloc((void **)&data, sizeof(hipfftComplex) * NX * BATCH);
    if(hipGetLastError() != hipSuccess) {
        std::cerr << "Cuda error: Failed to allocate." << std::endl;
        return -1;
    }
    hipMemcpy(data, data_h,  sizeof(hipfftComplex) * NX * BATCH, hipMemcpyHostToDevice);
    if(hipfftPlan1d(&plan, NX, HIPFFT_C2C, BATCH) != HIPFFT_SUCCESS) {
        std::cerr << "CUFFT error: Plan creation failed." << std::endl;
        return -1;
    }
    if(hipfftExecC2C(plan, data, data, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
        std::cerr << "CUFFT error: ExecC2C Forward failed." << std::endl;
        return -1;
    }
    if(hipDeviceSynchronize() != hipSuccess) {
        std::cerr << "Cuda error: Failed to synchronize." << std::endl;
        return -1;
    }
    hipMemcpy(data_h, data, NX * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

    for(int i = 0; i < NX; i++)
    {
        std::cout << data_h[i].x << "\t" << data_h[i].y << std::endl;
        fout << data_h[i].x << "\t" << data_h[i].y << std::endl;
    }

    hipfftDestroy(plan);
    hipFree(data);
    free(data_h);
    fin.close();
    fout.close();
    return 0;
}