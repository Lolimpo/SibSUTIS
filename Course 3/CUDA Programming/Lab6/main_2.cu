
#include <hip/hip_runtime.h>
#include <stdio.h>

void copying(float *h_a, float *h_b, float *d, unsigned int n, char *desc)
{
    printf("\n%s transfer\n", desc);
    unsigned int bytes = n * sizeof(float);
    float finishTime;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    hipMemcpy(d, h_a, bytes, hipMemcpyHostToDevice);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&finishTime, start, stop);
    printf("Host to Device time: %f\n", finishTime);
    printf("Host to Device bandwidth (GB/s): %f\n", bytes * 1e-6 / finishTime);

    hipEventRecord(start, 0);
    hipMemcpy(h_b, d, bytes, hipMemcpyDeviceToHost);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&finishTime, start, stop);
    printf("Device to Host time: %f\n", finishTime);
    printf("Device to Host bandwidth (GB/s): %f\n", bytes * 1e-6 / finishTime);

    for (int i = 0; i < n; i++)
    {
        if (h_a[i] != h_b[i])
        {
            printf("Smth failed :C\n");
            break;
        }
    }
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

int main()
{
    const int size = 4 * 1024 * 1024;
    const unsigned int bytes = size * sizeof(float);
    float *h_aPagable, *h_bPagable;
    float *h_aPinned, *h_bPinned;
    float *device;

    h_aPagable = (float *)malloc(bytes);
    h_bPagable = (float *)malloc(bytes);
    hipHostMalloc((void **)&h_aPinned, bytes, hipHostMallocDefault);
    hipHostMalloc((void **)&h_bPinned, bytes, hipHostMallocDefault);
    hipMalloc((void **)&device, bytes);

    for (int i = 0; i < size; i++)
        h_bPagable[i] = i;
    memcpy(h_aPinned, h_aPagable, bytes);
    memset(h_bPagable, 0, bytes);
    memset(h_bPinned, 0, bytes);

    copying(h_aPagable, h_bPagable, device, size, "Pageable");
    copying(h_aPinned, h_bPinned, device, size, "Pinned");

    hipFree(device);
    hipHostFree(h_aPinned);
    hipHostFree(h_bPinned);
    free(h_aPagable);
    free(h_bPagable);

    return 0;
}