
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void sum(const int *a, const int *b, int *c)
{
    int elemInd = threadIdx.x + blockDim.x * blockIdx.x;
    c[elemInd] = a[elemInd] + b[elemInd];
}

__global__ void multiply(int *a, int *b, int *c)
{
    int elemInd = threadIdx.x + blockDim.x * blockIdx.x;
    a[elemInd] *= b[elemInd];
    int i = gridDim.x * blockDim.x / 2;
    __syncthreads();
    while (i)
    {
        if (elemInd < i)
            a[elemInd] += a[elemInd + i];
        __syncthreads();
        i /= 2;
    }
    if (elemInd == 0)
        *c = a[0];
}

int main()
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    const int size = 1 << 20;
    float elapsedTime;
    int *hostArr, *stream0Arr, *stream1Arr, *hostResult, *stream0Result, *stream1Result;
    hipStream_t stream0, stream1;
    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);

    hipHostAlloc((void**)&hostArr, size * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&hostResult, size * sizeof(int), hipHostMallocDefault);
    for (int i = 0; i < size; ++i)
        hostArr[i] = rand() % 100;

    printf("sumTime1\tsumTime2\n");
    for (int i = 256; i < (size >> 1); i <<= 1)
    {
        printf ("i = %d\n", i);
        hipMalloc((void **) &stream0Arr, i * sizeof (int));
        hipMalloc((void **) &stream0Result, i * sizeof (int));
        hipMalloc((void **) &stream1Arr, i * sizeof (int));
        hipMalloc((void **) &stream1Result, i * sizeof (int));
        //1
        hipEventRecord (start, 0);
        for (int j = 0; j < size; j += 2 * i)
        {
            hipMemcpyAsync (stream0Arr, hostArr + j, sizeof (int) * i, hipMemcpyHostToDevice, stream0);
            sum <<<i / 256, 256, 0, stream0 >>> (stream0Arr, stream0Arr, stream0Result);
            hipMemcpyAsync (hostResult + j, stream0Result, sizeof (int) * i, hipMemcpyDeviceToHost, stream0);

            hipMemcpyAsync (stream1Arr, hostArr + j + i, sizeof (int) * i, hipMemcpyHostToDevice, stream1);
            sum <<<i / 256, 256, 0, stream1>>> (stream1Arr, stream1Arr, stream1Result);
            hipMemcpyAsync (hostResult + j + i, stream1Result, sizeof (int) * i, hipMemcpyDeviceToHost, stream1);
        }
        hipStreamSynchronize (stream0);
        hipStreamSynchronize (stream1);
        hipEventRecord (stop, 0);
        hipEventSynchronize (stop);
        hipEventElapsedTime (&elapsedTime, start, stop);
        printf ("%f\t", elapsedTime);

        memset (hostResult, 0, size * sizeof (int));
        //2
        hipEventRecord (start, 0);
        for (int j = 0; j < size; j += 2 * i)
        {
            hipMemcpyAsync (stream0Arr, hostArr + j, sizeof (int) * i, hipMemcpyHostToDevice, stream0);
            hipMemcpyAsync (stream0Arr, hostArr + j + i, sizeof (int) * i, hipMemcpyHostToDevice, stream1);
            sum <<<i / 256, 256, 0, stream0 >>> (stream0Arr, stream0Arr, stream0Result);
            sum <<<i / 256, 256, 0, stream1 >>> (stream1Arr, stream1Arr, stream1Result);
            hipMemcpyAsync (hostResult + j, stream0Result, sizeof (int) * i, hipMemcpyDeviceToHost, stream0);
            hipMemcpyAsync (hostResult + j + i, stream1Result, sizeof (int) * i, hipMemcpyDeviceToHost, stream1);
        }
        hipStreamSynchronize (stream0);
        hipStreamSynchronize (stream1);
        hipEventRecord (stop, 0);
        hipEventSynchronize (stop);
        hipEventElapsedTime (&elapsedTime, start, stop);
        printf ("%f\n", elapsedTime);

        memset(hostResult, 0, size * sizeof(int));

        hipFree(stream0Arr);
        hipFree(stream0Result);
        hipFree(stream1Arr);
        hipFree(stream1Result);
    }

    printf("multiplyTime1\tmultiplyTime2\n");
    for (int i = 256; i < (size >> 1); i <<= 1)
    {
        printf ("i = %d\n", i);
        hipMalloc ((void **) &stream0Arr, i * sizeof (int));
        hipMalloc ((void **) &stream0Result, i * sizeof (int));
        hipMalloc ((void **) &stream1Arr, i * sizeof (int));
        hipMalloc ((void **) &stream1Result, i * sizeof (int));
        //1
        hipEventRecord(start, 0);
        for (int j = 0; j < size; j += 2 * i)
        {
            hipMemcpyAsync(stream0Arr, hostArr + j, sizeof(int) * i, hipMemcpyHostToDevice, stream0);
            multiply <<<i / 256, 256, 0, stream0 >>> (stream0Arr, stream0Arr, stream0Result);
            hipMemcpyAsync(hostResult + j, stream0Result, sizeof(int) * i, hipMemcpyDeviceToDevice, stream0);

            hipMemcpyAsync(stream1Arr, hostArr + j + i, sizeof(int) * i, hipMemcpyHostToDevice, stream1);
            multiply <<<i / 256, 256, 0, stream1 >>> (stream1Arr, stream1Arr, stream1Result);
            hipMemcpyAsync(hostResult + j + i, stream1Result, sizeof(int) * i, hipMemcpyDeviceToDevice, stream1);
        }
        hipStreamSynchronize(stream0);
        hipStreamSynchronize(stream1);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop);
        printf("%f\t", elapsedTime);

        memset(hostResult, 0, size * sizeof(int));
        //2
        hipEventRecord(start, 0);
        for (int j = 0; j < size; j += 2 * i)
        {
            hipMemcpyAsync(stream0Arr, hostArr + j, sizeof(int) * i, hipMemcpyHostToDevice, stream0);
            hipMemcpyAsync(stream1Arr, hostArr + j + i, sizeof(int) * i, hipMemcpyHostToDevice, stream1);
            multiply <<<i / 256, 256, 0, stream0 >>> (stream0Arr, stream0Arr, stream0Result);
            multiply <<<i / 256, 256, 0, stream1 >>> (stream1Arr, stream1Arr, stream1Result);
            hipMemcpyAsync(hostResult + j, stream0Result, sizeof(int) * i, hipMemcpyDeviceToDevice, stream0);
            hipMemcpyAsync(hostResult + j + i, stream1Result, sizeof(int) * i, hipMemcpyDeviceToDevice, stream1);
        }
        hipStreamSynchronize(stream0);
        hipStreamSynchronize(stream1);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop);
        printf("%f\n", elapsedTime);

        memset(hostResult, 0, size * sizeof(int));

        hipFree(stream0Arr);
        hipFree(stream0Result);
        hipFree(stream1Arr);
        hipFree(stream1Result);
    }
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipHostFree(hostArr);
    hipHostFree(hostResult);
    return 0;
}
