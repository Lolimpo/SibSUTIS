#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define _USE_MATH_DEFINES
#include <math.h>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>

struct func
{
	float u, t, h;
	func(float _u, float _t, float _h) :
		u(_u), t(_t), h(_h) {};

	__host__ __device__
		float operator() (const float &x, const float &y) const 
		{
			return x + (y - x) * u * t / h;
		}
};

__global__ void funcCUDA(float *x, float *y, float u, float t, float h)
{
	int offset = threadIdx.x + blockDim.x * blockIdx.x;
	y[offset + 1] = x[offset + 1] + (x[offset] - x[offset + 1]) * u * t / h;
}

int main()
{
	float elapsedTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);
	thrust::host_vector<float> A_host(1 << 10);
	for (int i = 0; i < A_host.size(); ++i)
		A_host[i] = exp(-powf((i / 100.0 - 4.5), 2)) * 100 /  (2 * sqrtf(2 * M_PI));
	thrust::device_vector<float> A(1 << 10);
	thrust::copy(A_host.begin(), A_host.end(), A.begin());
	for (int i = 0; i < 1000; ++i)
		thrust::transform(A.begin() + 1, A.end(), A.begin(), A.begin(), func(1.1, 0.9, 1.4));
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("%f\n", elapsedTime);

	hipEventRecord(start, 0);
	float *vect, *vectRes, *vect_GPU, *vectRes_GPU;
	vect = (float*)malloc((1 << 10) * sizeof(float));
	vectRes = (float*)malloc((1 << 10) * sizeof(float));
	hipMalloc((void**)&vect_GPU, ((1 << 10) + 1) * sizeof(float));
	hipMalloc((void**)&vectRes_GPU, ((1 << 10) + 1) * sizeof(float));
	for (int i = 0; i < (1 << 10); ++i)
		vect[i] = exp(-powf((i / 100.0 - 4.5), 2)) * 100 / (2 * sqrtf(2 * M_PI));
	hipMemcpy(vect_GPU, vect, (1 << 10) * sizeof(float), hipMemcpyHostToDevice);
	for (int i = 0; i < 1000; ++i) 
	{
		funcCUDA << <4, 256 >> > (vect_GPU, vectRes_GPU, 1.1, 0.9, 1.4);
		hipMemcpy(vect_GPU, vectRes_GPU + 1, (1 << 10) * sizeof(float), hipMemcpyDeviceToDevice);
	}
	hipMemcpy(vectRes, vectRes_GPU + 1, (1 << 10) * sizeof(float), hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("%f\n", elapsedTime);
}

