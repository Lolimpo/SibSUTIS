
#include <hip/hip_runtime.h>
#include <iostream>

__global__
void gInitVector(float *vec, int n)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if(i < n)
        vec[i] = 1000.;
}

int main()
{
    float *A;
    int n;
    std::cin >> n;
    int blockSize = 128;
    int numBlocks = (n + blockSize - 1) / blockSize;
    A = new float [n];

    hipMallocManaged(&A, n * sizeof(float));
    gInitVector<<<numBlocks, blockSize>>>(A, n);
    hipDeviceSynchronize();
    hipFree(A);

    return 0;
}