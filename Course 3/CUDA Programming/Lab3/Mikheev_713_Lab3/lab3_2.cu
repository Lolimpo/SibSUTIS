#include <iostream>
#include <cmath>

#include <hip/hip_runtime.h>

__global__
void matrix_init(float* X)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int N = blockDim.x * gridDim.x;

    X[i + j * N] = float(i + j * N);
}

__global__
void matrix_init2(float *X)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int N = blockDim.x * gridDim.x;

    X[i] = float(i * N);
}
__global__
void transponse0(float* X, float *X_t)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int N = blockDim.x * gridDim.x;

    X_t[j + i * N] = X[i + j * N];
}

int main(int argc, char* argv[])
{
    float *A, *A_CUDA, *A_CUDA_T;
    int n = 2048;
    int numThreads = 32;
    int blockNum = n / numThreads;
    A = new float[n * n];
    A_CUDA = new float[n * n];
    A_CUDA_T = new float[n * n];

    hipMallocManaged(&A, n * n * sizeof(float));
    hipMallocManaged(&A_CUDA, n * n * sizeof(float));
    hipMallocManaged(&A_CUDA_T, n * n * sizeof(float));

    matrix_init<<<dim3(blockNum, blockNum), dim3(numThreads, numThreads)>>>(A_CUDA);
    hipDeviceSynchronize();
    matrix_init2<<<blockNum * blockNum, numThreads * numThreads>>>(A_CUDA);
    hipDeviceSynchronize();
    transponse0<<<dim3(blockNum, blockNum), dim3(numThreads, numThreads)>>>(A_CUDA, A_CUDA_T);
    hipDeviceSynchronize();
    
    hipFree(A_CUDA_T);
    hipFree(A_CUDA);
    hipFree(A);
    return 0;
}
