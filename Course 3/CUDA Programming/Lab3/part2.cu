#include <iostream>
#include <cmath>

#include <hip/hip_runtime.h>

__global__ void matrixInitByX(float *X) 
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int l = blockDim.x * gridDim.x;
    X[i + j * l] = (float) (threadIdx.x + blockDim.y * blockIdx.x);
}

__global__ void matrixInitByY(float *X) 
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int J = blockDim.x * gridDim.x;
    X[i + j * J] = (float) (threadIdx.y + blockDim.x * blockDim.y);
}

__global__ void transponse0(float *X, float *X_t)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int N = blockDim.x * gridDim.x;

    X_t[j + i * N] = X[i + j * N];
}

int main(int argc, char* argv[])
{
    float *A_CUDA, *A_CUDA2, *A_CUDA_T;
    int n = 1 << 10;
    std::cout << n << std::endl;
    int numThreads = 32;
    int blockNum = n / numThreads;
    hipMalloc((void **) &A_CUDA, n * n * sizeof(float));
    hipMalloc((void **) &A_CUDA2, n * n * sizeof(float));
    hipMallocManaged(&A_CUDA_T, n * n * sizeof(float));

    matrixInitByX<<<dim3(blockNum, blockNum), dim3(numThreads, numThreads)>>>(A_CUDA);
    hipDeviceSynchronize();
    matrixInitByY<<<dim3(blockNum, blockNum), dim3(numThreads, numThreads)>>>(A_CUDA2);
    hipDeviceSynchronize();
    transponse0<<<dim3(blockNum, blockNum), dim3(numThreads, numThreads)>>>(A_CUDA, A_CUDA_T);
    hipDeviceSynchronize();
    
    hipFree(A_CUDA);
    hipFree(A_CUDA2);
    hipFree(A_CUDA_T);
    return 0;
}