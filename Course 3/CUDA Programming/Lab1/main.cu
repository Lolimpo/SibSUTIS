#include <iostream>
#include <cmath>

#include <hip/hip_runtime.h>

#define CUDA_CHECK_RETURN(value) {\
cudaError_t _m_cudaStat = value;\
if (_m_cudaStat != hipSuccess) {\
 fprintf(stderr, "Error %s at line %d in file %s\n",\
 hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);\
 exit(1);\
} }


__global__
void compute(float X[], float Y[])
{
    X[threadIdx.x+blockDim.x*blockIdx.x] += Y[threadIdx.x+blockDim.x*blockIdx.x];
}

__global__
void matrix_init(float* X)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int N = blockDim.x * gridDim.x;

    X[i + j * N] = float(i + j * N);
}

__global__
void transponse0(float* X, float *X_t)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int N = blockDim.x * gridDim.x;

    X_t[j + i * N] = X[i + j * N];
}

int main(int argc, char *argv[])
{
    float *X, *Y, *Z, *CUDA_X, *CUDA_Y;
    for(int n = pow(2, 28), i = 0; i < 19; n /= 2, i++)
    {
    //int n = pow(2, 10);
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        float milliseconds = 0.0;
        int thNum = std::stoi(argv[1]);
        int blNum = n / thNum;

        X = new float[n];
        Y = new float[n];
        Z = new float[n];
        for(int j = 0; j < n; j++)
        {
            X[j] = (float) rand () / RAND_MAX;
            Y[j] = (float) rand () / RAND_MAX;
        }
        CUDA_CHECK_RETURN(hipMalloc((void**) &CUDA_X, n * sizeof (float)));
        CUDA_CHECK_RETURN(hipMalloc((void**) &CUDA_Y, n * sizeof (float)));
        CUDA_CHECK_RETURN(hipMemcpy(CUDA_X, X, n * sizeof (float), hipMemcpyHostToDevice));
        CUDA_CHECK_RETURN(hipMemcpy(CUDA_Y, Y, n * sizeof (float), hipMemcpyHostToDevice));
        
        hipEventRecord (start);
        compute<<<dim3(blNum), dim3(thNum)>>> (CUDA_X, CUDA_Y);
        CUDA_CHECK_RETURN(hipDeviceSynchronize());

        hipEventRecord (stop);
        CUDA_CHECK_RETURN(hipMemcpy (Z, CUDA_X, n * sizeof (float), hipMemcpyDeviceToHost));
        hipEventSynchronize (stop);
        hipEventElapsedTime (&milliseconds, start, stop);
        std::cout << milliseconds << std::endl;

        for (int j = 0; j < n; j++)
        {
            if (Z[j] != X[j] + Y[j])
            {
                std::cout << "Some big mess happened :c" << std::endl;
                return 0;
            }
        }
        hipFree(CUDA_X);
        hipFree(CUDA_Y);
        delete[] X;
        delete[] Y;
        delete[] Z;
    }
    return 0;
}
