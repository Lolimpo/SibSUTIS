#include <iostream>

#include <hip/hip_runtime.h>

void Output(float *a, int N) {
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++)
            fprintf(stdout, "%g\t", a[j + i * N]);
        fprintf(stdout, "\n");
    }
    fprintf(stdout, "\n\n\n");
}

__global__ void gInitializeStorage(float *storage_d) {
    unsigned i = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned j = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned N = blockDim.x * gridDim.x;
    storage_d[i + j * N] = (float) (i + j * N);
}

__global__ void gTranspose0(const float *storage_d, float *storage_d_t) {
    unsigned i = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned j = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned N = blockDim.x * gridDim.x;
    storage_d_t[j + i * N] = storage_d[i + j * N];
}

__global__ void gTranspose11(const float *storage_d, float *storage_d_t) {
    extern __shared__ float buffer[];
    unsigned i = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned j = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned N = blockDim.x * gridDim.x;
    buffer[threadIdx.y + threadIdx.x * blockDim.y] = storage_d[i + j * N];
    __syncthreads();
    i = threadIdx.x + blockIdx.y * blockDim.x;
    j = threadIdx.y + blockIdx.x * blockDim.y;
    storage_d_t[i + j * N] = buffer[threadIdx.x + threadIdx.y * blockDim.x];
}

#define SH_DIM 32

__global__ void gTranspose12(const float *storage_d, float *storage_d_t) {
    __shared__ float buffer_s[SH_DIM][SH_DIM];
    unsigned i = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned j = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned N = blockDim.x * gridDim.x;
    buffer_s[threadIdx.y][threadIdx.x] = storage_d[i + j * N];
    __syncthreads();
    i = threadIdx.x + blockIdx.y * blockDim.x;
    j = threadIdx.y + blockIdx.x * blockDim.y;
    storage_d_t[i + j * N] = buffer_s[threadIdx.x][threadIdx.y];
}

__global__ void gTranspose2(const float *storage_d, float *storage_d_t) {
    __shared__ float buffer[SH_DIM][SH_DIM + 1];
    unsigned i = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned j = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned N = blockDim.x * gridDim.x;
    buffer[threadIdx.y][threadIdx.x] = storage_d[i + j * N];
    __syncthreads();
    i = threadIdx.x + blockIdx.y * blockDim.x;
    j = threadIdx.y + blockIdx.x * blockDim.y;
    storage_d_t[i + j * N] = buffer[threadIdx.x][threadIdx.y];
}

int main(int argc, char *argv[]) {
    if (argc < 3) {
        fprintf(stderr, "USAGE: matrix <dimension of matrix><dimension_of_threads>\n");
        return -1;
    }
    int N = atoi(argv[1]);
    int dim_of_threads = atoi(argv[2]);
    if (N % dim_of_threads) {
        fprintf(stderr, "change dimensions\n");
        return -1;
    }
    int dim_of_blocks = N / dim_of_threads;
    const int max_size = 1 << 8;
    if (dim_of_blocks > max_size) {
        fprintf(stderr, "too many blocks\n");
        return -1;
    }

    float *storage_d, *storage_d_t, *storage_h;
    hipMalloc((void **) &storage_d, N * N * sizeof(float));
    hipMalloc((void **) &storage_d_t, N * N * sizeof(float));
    storage_h = (float *) calloc(N * N, sizeof(float));

    gInitializeStorage<<<dim3(dim_of_blocks, dim_of_blocks),
    dim3(dim_of_threads, dim_of_threads)>>>(storage_d);
    hipDeviceSynchronize();
    memset(storage_h, 0.0, N * N * sizeof(float));
    hipMemcpy(storage_h, storage_d, N * N * sizeof(float), hipMemcpyDeviceToHost);
//    Output(storage_h, N);

    gTranspose0<<<dim3(dim_of_blocks, dim_of_blocks),
    dim3(dim_of_threads, dim_of_threads)>>>(storage_d, storage_d_t);
    hipDeviceSynchronize();
    memset(storage_h, 0.0, N * N * sizeof(float));
    hipMemcpy(storage_h, storage_d_t, N * N * sizeof(float), hipMemcpyDeviceToHost);
//    Output(storage_h, N);

    gTranspose11<<<dim3(dim_of_blocks, dim_of_blocks),
    dim3(dim_of_threads, dim_of_threads),
    dim_of_threads*dim_of_threads*sizeof(float)>>>(storage_d, storage_d_t);
    hipDeviceSynchronize();
    memset(storage_h, 0.0, N * N * sizeof(float));
    hipMemcpy(storage_h, storage_d_t, N * N * sizeof(float), hipMemcpyDeviceToHost);
//    Output(storage_h, N);

    gTranspose12<<<dim3(dim_of_blocks, dim_of_blocks),
    dim3(dim_of_threads, dim_of_threads)>>>(storage_d, storage_d_t);
    hipDeviceSynchronize();
    memset(storage_h, 0.0, N * N * sizeof(float));
    hipMemcpy(storage_h, storage_d_t, N * N * sizeof(float), hipMemcpyDeviceToHost);
//    Output(storage_h, N);

    gTranspose2<<<dim3(dim_of_blocks, dim_of_blocks),
    dim3(dim_of_threads, dim_of_threads)>>>(storage_d, storage_d_t);
    hipDeviceSynchronize();
    memset(storage_h, 0.0, N * N * sizeof(float));
    hipMemcpy(storage_h, storage_d_t, N * N * sizeof(float), hipMemcpyDeviceToHost);
//    Output(storage_h, N);

    hipFree(storage_d);
    hipFree(storage_d_t);
    free(storage_h);
    return 0;
}